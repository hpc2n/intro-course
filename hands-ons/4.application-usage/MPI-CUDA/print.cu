#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

 
 __global__ void __print_kernel__ ()
 {
    printf("GPU says, Hello world! \n");
 }
 
 extern "C" void print_gpu()
{
int nDevices; 

     hipGetDeviceCount(&nDevices);
     printf("Nr. GPUs %d \n", nDevices);

     __print_kernel__ <<<1,1>>> ();

     hipDeviceSynchronize();
}
